/*
 * Copyright (c) 2008-2010
 *	Takao, Yasuyoshi and Nakata, Maho
 * 	All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR AND CONTRIBUTORS ``AS IS'' AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHOR OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS
 * OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
 * HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
 * OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGE.
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

void __attribute__ ((constructor)) SetDevice(void);

void SetDevice()
{
    int gpudevice = 0;  // cpu number 0,1,...
        
    int device_count=0;
    int device;

    printf("Initialize device...\n");
    
    hipGetDeviceCount( &device_count);
    printf("device_count : %d\n", device_count);

    hipError_t cudareturn;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,  gpudevice);

    printf("device name -> %s \n", deviceProp.name);

    if (deviceProp.warpSize <= 1){
        printf("warning, CUDA Device Emulation (CPU)  detected, exiting\n");
        exit(1);
    }
    //set GPU that is to be used
    cudareturn=hipSetDevice(gpudevice);
    printf("cudareturn -> %d\n", cudareturn);

    if (cudareturn == hipErrorInvalidDevice){
        perror("hipSetDevice returned  hipErrorInvalidDevice");
    }else{
        hipGetDevice(&device);
        printf("cudaGetDevice()=%d\n" ,device);
    }

    //dummy memory allocation for initialization
    double *Dummyd;
    hipMalloc((void **)&Dummyd, 16);
    hipFree(Dummyd);
}
